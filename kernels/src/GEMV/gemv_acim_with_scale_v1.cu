#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:49:00 on Mon, Oct 09, 2023
//
// Description: warp8 smem hgemv

#include "GEMV/common_gemv.h"
#include "GEMV/gemv_util.h"

template <typename T> __device__ int get_bit_(const T in_data, const int bit_pos) {
  // printf("in_data: %d\n", in_data);
  int temp = (in_data) >> (bit_pos);
  return temp & 0b1;
}

__device__ int adaptive_quantize_(const int in_data, const int bitwidth) {
  int max_value;
  int val;

  max_value = (1 << bitwidth) - 1;

  int offset = in_data ? 0 : 1;
  int div = CEIL_DIV(in_data, max_value) + offset;
  div--;
  div |= div >> 1;
  div |= div >> 2;
  div |= div >> 4;
  div |= div >> 8;
  div |= div >> 16;
  div++;

  val = ROUND_DIV(in_data, div) * div;
  return val;
}

template <int IN_BW, int W_BW>
__global__ void gemv_acim_with_scale_kernel_v1(const char *__restrict__ A,
                                               const char *__restrict__ B, float *__restrict__ C,
                                               const int M, const int N, const int K,
                                               const float *in_scale, const float *weight_scale,
                                               const bool quant, const int sm_offset) {

  // printf("hi");

  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int tid = ty * blockDim.x + tx;
  // const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int scale_per_row = K / QUANT_GROUP_SIZE;

  extern __shared__ char shared_mem[];

  char *Asub = (char *)shared_mem;
  float *psum_shared = (float *)((char *)shared_mem + sm_offset);
  // __shared__ char Asub[KMAX];
  // __shared__ float psum_shared[KMAX / WK];

  int bp_psum[IN_BW][W_BW]; // 64 int per thread. this merge with warp level primitives
  char in_bp[IN_BW];        // each bit planes. layer used when outer product
  char wg_bp[W_BW];         // same

  // load A into Asub with all threads in a block
  // TODO use float4
  const int load_per_thread = K / THREAD_NUMS(K);
  int load_iter = 0;
  do {
    int kidx = THREAD_NUMS(K) * load_iter + blockDim.x * ty + tx;
    Asub[kidx] = A[kidx];
    load_iter++;
  } while (load_iter < load_per_thread);

  __syncthreads();

  // loop over k and n dimension
  const int k_iter = WK / WARP_SIZE;
  const int n_iter = WN;
  int warp_xidx = tx / WARP_SIZE;
  int warp_yidx = ty;
  int warp_idx = warp_yidx * WARP_X_NUM(K) + warp_xidx;

#pragma unroll
  for (int i = 0; i < IN_BW; i++) {
#pragma unroll
    for (int j = 0; j < W_BW; j++) {
      bp_psum[i][j] = 0;
    }
  }

  // if ((tid % WARP_SIZE) == 0) {
  //   psum_shared[warp_idx] = 0;
  //   if (blockIdx.y == 0) {
  //     printf("tx: %d, warp_xidx: %d, warp_yidx: %d, psum_shared: %f\n", tx, warp_xidx, warp_yidx,
  //            psum_shared[warp_xidx]);
  //   }
  // }

  // if ((tid < K) && IN_BW == 4 && W_BW == 4) {
  //   C[tid] = psum_shared[0];
  // }

  for (int n = 0; n < n_iter; n++) {
    int nidx = by * TN + warp_yidx * WN + n;
    // if (tid % WARP_SIZE == 0) {
    //   psum_shared[warp_idx] = 0;
    // if (blockIdx.y == 0) {
    // printf("tx: %d, warp_xidx: %d, warp_yidx: %d, psum_shared: %f\n", tx, warp_xidx, warp_yidx,
    //        psum_shared[warp_xidx]);
    // }
    // }
    psum_shared[warp_idx] = 0;
    for (int k = 0; k < k_iter; k++) {

      int kidx_offset_in_warp = k * WARP_SIZE;
      int kidx = warp_xidx * WK + kidx_offset_in_warp + tx % WARP_SIZE;
      bool is_boundary =
          ((DOWN_TO_MULTIPLE(kidx_offset_in_warp, WARP_SIZE) + WARP_SIZE) % QUANT_GROUP_SIZE == 0);
      // int is_outlier_region = ((kidx + QUANT_GROUP_SIZE) > K);
      // int bitwidth = is_outlier_region ? NOM_BW : MAX_BW;

// parsing each bit plane of A and B
#pragma unroll
      for (int i = 0; i < IN_BW; i++) {
        in_bp[i] = get_bit_<char>(Asub[kidx], i);
      }

#pragma unroll
      for (int i = 0; i < W_BW; i++) {
        wg_bp[i] = get_bit_<char>(B[nidx * K + kidx], i);
      }

      // compute the partial sum
#pragma unroll
      for (int wgb = 0; wgb < W_BW; wgb++) {
        // char wg_bp = get_bit_<char>(B[nidx * K + kidx], wgb);
#pragma unroll
        for (int inb = 0; inb < IN_BW; inb++) {
          // char in_bp = get_bit_<char>(Asub[kidx], inb);
          bp_psum[inb][wgb] += (in_bp[inb] * wg_bp[wgb]);
          // bp_psum[inb][wgb] += (in_bp * wg_bp);
          if (blockIdx.y == 0 && n == 0 && tx / WARP_SIZE == 0 && inb == 0 && wgb == 0 &&
              warp_yidx == 0) {
            printf("compute psum in reg. kidx: %d, inb: %d, wgb: %d, psum: %d\n", kidx, inb, wgb,
                   bp_psum[inb][wgb]);
          }
        }
      }

      if (is_boundary) {
        // if (blockIdx.y == 0 && n == 0 && tx % 32 == 0) {
        // printf("boundary enterd, kidx: %d\n", kidx);
        // }
        // reduce across the threads in a warp
        constexpr unsigned int mask = 0xffffffff;
#pragma unroll
        for (size_t i = WARP_SIZE / 2; i >= 1; i /= 2) {
#pragma unroll
          for (int inb = 0; inb < IN_BW; inb++) {
#pragma unroll
            for (int wgb = 0; wgb < W_BW; wgb++) {
              bp_psum[inb][wgb] += __shfl_xor_sync(mask, bp_psum[inb][wgb], i);
              // if (inb == 0 && wgb == 0 && blockIdx.y == 0 && n == 0 && tx == 0) {
              //   printf("reduce across threads, kidx: %d, psum: %d\n", kidx, bp_psum[inb][wgb]);
              // }
            }
          }
        }

        // quantize the partial sum
        if (tid % WARP_SIZE == 0) { // master warp
          int result = 0;
          if (quant) {
#pragma unroll
            for (int inb = 0; inb < IN_BW; inb++) {
#pragma unroll
              for (int wgb = 0; wgb < W_BW; wgb++) {
                bp_psum[inb][wgb] = adaptive_quantize_(bp_psum[inb][wgb], ADC_BITWIDTH);
              }
            }
          }

          for (int inb = 0; inb < IN_BW; inb++) {
            for (int wgb = 0; wgb < W_BW; wgb++) {
              bp_psum[inb][wgb] = (inb == (IN_BW - 1)) ? -bp_psum[inb][wgb] : bp_psum[inb][wgb];
              bp_psum[inb][wgb] = (wgb == (W_BW - 1)) ? -bp_psum[inb][wgb] : bp_psum[inb][wgb];
              result += (bp_psum[inb][wgb] << (inb + wgb));
            }
          }

          // accumulate the partial sum
          float curr_in_scale = in_scale[kidx / QUANT_GROUP_SIZE];
          float curr_weight_scale = weight_scale[nidx * scale_per_row + kidx / QUANT_GROUP_SIZE];
          psum_shared[warp_idx] += result * curr_in_scale * curr_weight_scale;
          // curr_weight_scale; if (blockIdx.y == 0 && n == 0) {
          //   printf("acccum to smem. kidx: %d, nidx : %d, smem_idx : %d, result : %d, is : %f, ws
          //   : "
          //          "%f, ps : %f\n",
          //          kidx, nidx, warp_idx, result, curr_in_scale, curr_weight_scale,
          //          psum_shared[warp_idx]);
          // }
        }

        // clear psum reg
#pragma unroll
        for (int i = 0; i < IN_BW; i++) {
#pragma unroll
          for (int j = 0; j < W_BW; j++) {
            bp_psum[i][j] = 0;
          }
        }
      }
    }
    // reduce across warps for psum_shared
    __syncthreads();

    volatile float *v_psum_shared = psum_shared;
    if (blockDim.x >= 1024 && tid < (512 / WARP_SIZE)) {
      v_psum_shared[tid] += v_psum_shared[tid + 512 / WARP_SIZE];
      // if (blockIdx.y == 0 && n == 0 && warp_yidx == 0) {
      //   printf("reduce across warp. iter 0. tx: %d, psum_shared: %f\n", tid, v_psum_shared[tid]);
      // }
    }
    if (blockDim.x >= 512 && tid < (256 / WARP_SIZE)) {
      // if (blockIdx.y == 0 && n == 0) {
      //   printf("reduce across warp. iter 1. tx: %d, psum_shared 1: %f pusm_shared 2: %f\n", tx,
      //          v_psum_shared[tx], v_psum_shared[tx + 256 / WARP_SIZE]);
      // }
      v_psum_shared[tid] += v_psum_shared[tid + 256 / WARP_SIZE];
      // if (blockIdx.y == 0 && n == 0) {
      //   printf("reduce across warp result. iter 1. tx: %d, psum_shared: %f\n", tx,
      //          v_psum_shared[tx]);
      // }
    }
    if (blockDim.x >= 256 && tid < (128 / WARP_SIZE)) {
      // if (blockIdx.y == 0 && n == 0) {
      //   printf("reduce across warp. iter 2. tx: %d, psum_shared 1: %f pusm_shared 2: %f\n", tx,
      //          v_psum_shared[tx], v_psum_shared[tx + 128 / WARP_SIZE]);
      // }
      v_psum_shared[tid] += v_psum_shared[tid + 128 / WARP_SIZE];
    }
    if (blockDim.x >= 128 && tid < (64 / WARP_SIZE)) {
      v_psum_shared[tid] += v_psum_shared[tid + 64 / WARP_SIZE];
      // if (blockIdx.y == 0 && n == 0) {
      //   printf("reduce across warp. iter 3. tx: %d, psum_shared: %f\n", tx, psum_shared[tx]);
      // }
    }
    if (blockDim.x >= 64 && tid < (32 / WARP_SIZE)) {
      v_psum_shared[tid] += v_psum_shared[tid + 32 / WARP_SIZE];
      // if (blockIdx.y == 0 && n == 0) {
      //   printf("reduce across warp. iter 4. tx: %d, psum_shared: %f\n", tx, psum_shared[tx]);
      // }
    }

    __syncthreads();
    // store the result
    if (tx == 0) {
      // C[nidx] += psum_shared[0];
      C[nidx] = psum_shared[0];
    }
  }
}

void gemv_acim_with_scale_v1(const char *A, const char *B, float *C, const int M, const int N,
                             const int K, const float *in_scale, const float *weight_scale,
                             const int input_bw, const int weight_bw, const bool quant) {
  // __glibcxx_assert(M == 1);
  __glibcxx_assert(K % WK == 0);
  __glibcxx_assert(K % QUANT_GROUP_SIZE == 0);
  __glibcxx_assert(WK % QUANT_GROUP_SIZE == 0);
  __glibcxx_assert(WK % WARP_SIZE == 0);
  __glibcxx_assert(KMAX % WK == 0);
  __glibcxx_assert(TN % WN == 0);
  __glibcxx_assert(N % TN == 0);

  static char *A_gpu, *B_gpu, *B_out_gpu;
  static float *C_gpu, *in_scale_gpu, *weight_scale_gpu, *weight_scale_out_gpu;
  static hipStream_t stream;
  static bool init = false;
  int sm_size;
  int sm_offset;

  // __shared__ char Asub[KMAX];
  // __shared__ float psum_shared[KMAX / WK];

  if (!init) {
    init = true;
    CHECK_CUDA(hipMalloc(&A_gpu, MKMAX * sizeof(char)));
    CHECK_CUDA(hipMalloc(&B_gpu, KNMAX * sizeof(char)));
    // CHECK_CUDA(hipMalloc(&B_out_gpu, KN_OUT_MAX * sizeof(char)));
    CHECK_CUDA(hipMalloc(&C_gpu, MNMAX * sizeof(float)));
    CHECK_CUDA(hipMalloc(&in_scale_gpu, (MKMAX / QUANT_GROUP_SIZE) * sizeof(float)));
    CHECK_CUDA(hipMalloc(&weight_scale_gpu, (KNMAX / QUANT_GROUP_SIZE) * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&weight_scale_out_gpu, (KN_OUT_MAX / QUANT_GROUP_SIZE) *
    // sizeof(float)));
    CHECK_CUDA(hipStreamCreate(&stream));
  }

  CHECK_CUDA(hipMemcpyAsync(B_gpu, B, K * N * sizeof(char), hipMemcpyHostToDevice, stream));
  CHECK_CUDA(hipMemcpyAsync(weight_scale_gpu, weight_scale,
                             (K / QUANT_GROUP_SIZE) * N * sizeof(float), hipMemcpyHostToDevice,
                             stream));
  // for (int i = 0; i < N; i++) {
  //   CHECK_CUDA(hipMemcpyAsync(B_gpu, &B[i * K], (K - QUANT_GROUP_SIZE) * sizeof(char),
  //                              hipMemcpyHostToDevice, stream));
  //   CHECK_CUDA(hipMemcpyAsync(weight_scale_gpu, &weight_scale[i * (K / QUANT_GROUP_SIZE)],
  //                              ((K - QUANT_GROUP_SIZE) / QUANT_GROUP_SIZE) * sizeof(float),
  //                              hipMemcpyHostToDevice, stream));

  //   CHECK_CUDA(hipMemcpyAsync(B_out_gpu, &B[i * K + (K - QUANT_GROUP_SIZE)],
  //                              QUANT_GROUP_SIZE * sizeof(char), hipMemcpyHostToDevice, stream));
  //   CHECK_CUDA(hipMemcpyAsync(
  //       weight_scale_out_gpu,
  //       &weight_scale[i * (K / QUANT_GROUP_SIZE) + (K - QUANT_GROUP_SIZE) / QUANT_GROUP_SIZE],
  //       sizeof(float), hipMemcpyHostToDevice, stream));
  // }

  dim3 norm_mm_block(BLOCK_SIZE_X(K), BLOCK_SIZE_Y);
  dim3 norm_mm_grid(GRID_SIZE_X, GRID_SIZE_Y(N));

  // dim3 out_mm_block(BLOCK_SIZE_X(QUANT_GROUP_SIZE), BLOCK_SIZE_Y);
  // dim3 out_mm_grid(GRID_SIZE_X, GRID_SIZE_Y(N));
  sm_size = K * sizeof(char) + WARP_X_NUM(K) * WARP_Y_NUM * sizeof(float);
  sm_offset = K * sizeof(char);

  for (int m = 0; m < M; m++) {
    const int cM = 1;
    CHECK_CUDA(
        hipMemcpyAsync(A_gpu, &A[m * K], cM * K * sizeof(char), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(in_scale_gpu, &in_scale[m * (K / QUANT_GROUP_SIZE)],
                               cM * (K / QUANT_GROUP_SIZE) * sizeof(float), hipMemcpyHostToDevice,
                               stream));

    // printf("block size : %d / %d, grid size : %d / %d\n", mm_block.x, mm_block.y, mm_grid.x,
    //        mm_grid.y);

    // normal region

    gemv_acim_with_scale_kernel_v1<8, 8><<<norm_mm_grid, norm_mm_block, sm_size, stream>>>(
        A_gpu, B_gpu, C_gpu, cM, N, K, in_scale_gpu, weight_scale_gpu, quant, sm_offset);

    // // outlier region
    // sm_size =
    //     QUANT_GROUP_SIZE * sizeof(char) + WARP_X_NUM(QUANT_GROUP_SIZE) * WARP_Y_NUM *
    //     sizeof(float);
    // sm_offset = QUANT_GROUP_SIZE * sizeof(char);

    // gemv_acim_with_scale_kernel_v1<8, 8><<<out_mm_grid, out_mm_block, sm_size, stream>>>(
    //     &A_gpu[K - QUANT_GROUP_SIZE], B_out_gpu, C_gpu, cM, N, QUANT_GROUP_SIZE,
    //     &in_scale_gpu[(K - QUANT_GROUP_SIZE) / QUANT_GROUP_SIZE], weight_scale_out_gpu, quant,
    //     sm_offset);

    CHECK_CUDA(hipMemcpy(&C[m * N], C_gpu, cM * N * sizeof(float), hipMemcpyDeviceToHost));
  }

  // CHECK_CUDA(hipFree(A_gpu));
  // CHECK_CUDA(hipFree(B_gpu));
  // CHECK_CUDA(hipFree(C_gpu));
  // CHECK_CUDA(hipFree(in_scale_gpu));
  // CHECK_CUDA(hipFree(weight_scale_gpu));
}